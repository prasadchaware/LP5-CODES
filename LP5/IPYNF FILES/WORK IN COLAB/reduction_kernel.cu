
#include <hip/hip_runtime.h>
// Parallel reduction for finding minimum value
__global__ void min_reduction(float *input, float *output, int n) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        sdata[tid] = input[i];
    else
        sdata[tid] = INFINITY;
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = min(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }
    if (tid == 0) output[blockIdx.x] = sdata[0];
}

// Parallel reduction for finding maximum value
__global__ void max_reduction(float *input, float *output, int n) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        sdata[tid] = input[i];
    else
        sdata[tid] = -INFINITY;
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = max(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }
    if (tid == 0) output[blockIdx.x] = sdata[0];
}

// Parallel reduction for finding sum
__global__ void sum_reduction(float *input, float *output, int n) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        sdata[tid] = input[i];
    else
        sdata[tid] = 0;
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) output[blockIdx.x] = sdata[0];
}

// Parallel reduction for finding average
__global__ void avg_reduction(float *input, float *output, int n) {
    extern __shared__ float sdata[];
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n)
        sdata[tid] = input[i];
    else
        sdata[tid] = 0;
    __syncthreads();
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
    if (tid == 0) output[blockIdx.x] = sdata[0] / n;
}
